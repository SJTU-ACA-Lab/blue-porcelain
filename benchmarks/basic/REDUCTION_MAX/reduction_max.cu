// Copyright 2022, ACALab of SJTU


#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

using namespace std;

const int threadsPerBlock = 512;
const int N = (1 << 11) - 3;
const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

__global__ void sum(int* arr, int* out, int N) {
  __shared__ int s_data[threadsPerBlock];
  unsigned int tid = threadIdx.x;
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < N) {
    s_data[tid] = arr[i];
  }
  __syncthreads();

  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s && i + s < N) {
      if (s_data[tid] < s_data[tid + s]) s_data[tid] = s_data[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0) {
    out[blockIdx.x] = s_data[0];
  }
}

int varifyOutput(int* predict, int* arr, int N) {
  int pred = -2000, result = -2000;
  for (int i = 0; i < blocksPerGrid; i++) {
    if (predict[i] > pred) pred = predict[i];
  }
  for (int i = 0; i < N; i++) {
    if (arr[i] > result) result = arr[i];
  }
  return pred != result;
}

int main() {
  int *a_host, *r_host;
  int *a_device, *r_device;

  hipHostMalloc(&a_host, N * sizeof(int), hipHostMallocDefault);
  hipHostMalloc(&r_host, blocksPerGrid * sizeof(int), hipHostMallocDefault);

  hipMalloc(&a_device, N * sizeof(int));
  hipMalloc(&r_device, blocksPerGrid * sizeof(int));

  std::random_device rd;
  std::default_random_engine eng(rd());
  std::uniform_int_distribution<int> distr(-1000, 1000);

  for (int i = 0; i < N; i++) {
    a_host[i] = distr(eng);
  }
  for (int i = 0; i < blocksPerGrid; i++) {
    r_host[i] = 0.0;
  }

  hipMemcpyAsync(a_device, a_host, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpyAsync(r_device, r_host, blocksPerGrid * sizeof(int),
                  hipMemcpyHostToDevice);

  sum<<<blocksPerGrid, threadsPerBlock, 0>>>(a_device, r_device, N);

  hipMemcpy(r_host, r_device, blocksPerGrid * sizeof(int),
             hipMemcpyDeviceToHost);

  int error = varifyOutput(r_host, a_host, N);

  hipFree(r_device);
  hipFree(a_device);
  hipHostFree(r_host);
  hipHostFree(a_host);

  if (error) {
    cout << "Calculation Error!\n";
    return 1;
  }
  cout << "Calculation right!\n";
  return 0;
}