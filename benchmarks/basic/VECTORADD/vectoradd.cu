// Copyright 2022, ACALab of SJTU


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 1024

__global__ void vecAddF(float *a, float *b, float *c) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < N) c[index] = a[index] + b[index];
}

__global__ void vecAddI32(int *a, int *b, int *c) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < N) c[index] = a[index] + b[index];
}

__global__ void vecAddI64(long long *a, long long *b, long long *c) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  c[index] = a[index] + b[index];
}

// basically just fills the array with index.
float getRandData(int min, int max) {
  float m1 = (double)(rand() % 101) / 101;
  min++;
  float m2 = (double)((rand() % (max - min + 1)) + min);
  m2 = m2 - 1;
  return m1 + m2;
}

void fill_array(float *data) {
  for (int i = 0; i < N; ++i) {
    data[i] = getRandData(0, 1000);
  }
}

void fill_array(int *data) {
  for (int idx = 0; idx < N; idx++) data[idx] = rand() % 200000;
}

void fill_array(long long *data) {
  for (int idx = 0; idx < N; idx++) data[idx] = INT_MAX + idx;
}

int main() {
  srand((unsigned)time(NULL));

  float *fa, *fb, *fc;
  float *d_fa, *d_fb, *d_fc;

  int *a, *b, *c;
  int *d_a, *d_b, *d_c;

  long long *la, *lb, *lc;
  long long *d_la, *d_lb, *d_lc;

  int threads_per_block = 0, no_of_blocks = 0;

  int fsize = N * sizeof(float);
  int size = N * sizeof(int);
  int lsize = N * sizeof(long long);

  // Alloc space for host copies of a, b, c and setup input values
  fa = (float *)malloc(fsize);
  fill_array(fa);
  fb = (float *)malloc(fsize);
  fill_array(fb);
  fc = (float *)malloc(fsize);

  a = (int *)malloc(size);
  fill_array(a);
  b = (int *)malloc(size);
  fill_array(b);
  c = (int *)malloc(size);

  la = (long long *)malloc(lsize);
  fill_array(la);
  lb = (long long *)malloc(lsize);
  fill_array(lb);
  lc = (long long *)malloc(lsize);

  // Alloc space for device copies of a, b, c
  hipMalloc((void **)&d_fa, fsize);
  hipMalloc((void **)&d_fb, fsize);
  hipMalloc((void **)&d_fc, fsize);

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  hipMalloc((void **)&d_la, lsize);
  hipMalloc((void **)&d_lb, lsize);
  hipMalloc((void **)&d_lc, lsize);

  // Copy inputs to device
  hipMemcpy(d_fa, fa, fsize, hipMemcpyHostToDevice);
  hipMemcpy(d_fb, fb, fsize, hipMemcpyHostToDevice);
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  hipMemcpy(d_la, la, lsize, hipMemcpyHostToDevice);
  hipMemcpy(d_lb, lb, lsize, hipMemcpyHostToDevice);

  threads_per_block = 256;
  no_of_blocks = (N + threads_per_block - 1) / threads_per_block;
  vecAddF<<<no_of_blocks, threads_per_block>>>(d_fa, d_fb, d_fc);
  vecAddI32<<<no_of_blocks, threads_per_block>>>(d_a, d_b, d_c);
  vecAddI64<<<no_of_blocks, threads_per_block>>>(d_la, d_lb, d_lc);

  // Copy result back to host
  hipMemcpy(fc, d_fc, fsize, hipMemcpyDeviceToHost);
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  hipMemcpy(lc, d_lc, lsize, hipMemcpyDeviceToHost);

  int flag = 0;
  for (int i = 0; i < N; i++) {
    if (fc[i] != fa[i] + fb[i]) {
      printf("FC[%d]:%f != %f + %f\n", i, fc[i], fa[i], fb[i]);
      flag = 1;
    }
    if (c[i] != a[i] + b[i]) {
      printf("C[%d]:%d != %d + %d\n", i, c[i], a[i], b[i]);
      flag = 1;
    }
    if (lc[i] != la[i] + lb[i]) {
      printf("LC[%d]:%lld != %lld + %lld\n", i, lc[i], la[i], lb[i]);
      flag = 1;
    }
  }

  free(fa);
  free(fb);
  free(fc);
  free(a);
  free(b);
  free(c);
  free(la);
  free(lb);
  free(lc);
  hipFree(d_fa);
  hipFree(d_fb);
  hipFree(d_fc);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipFree(d_la);
  hipFree(d_lb);
  hipFree(d_lc);

  if (flag == 0) {
    printf("Calculation right!\n");
    return 0;
  }

  return 1;
}
