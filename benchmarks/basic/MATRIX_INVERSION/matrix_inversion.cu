// Copyright 2022, ACALab of SJTU


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#include <fstream>
#include <iostream>
#include <string>
#include <vector>

using namespace std;

#define blocksize 8

/*storing matrix*/
void matrix_read(float *L, int dimension) {
  FILE *fp;
  int row, col;

  fp = fopen("randomMatrix_100.input", "r");  // open output file
  if (fp == NULL)                             // open failed
    return;

  for (row = 0; row < dimension; row++) {
    for (col = 0; col < dimension; col++)
      if (fscanf(fp, "%f,", &L[row * dimension + col]) == EOF)
        break;  // read data

    if (feof(fp)) break;  // if the file is over
  }

  fclose(fp);  // close file
}

__global__ void nodiag_normalize(float *A, float *I, int n, int i) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < n && y < n)
    if (x == i && x != y) {
      I[x * n + y] /= A[i * n + i];
      A[x * n + y] /= A[i * n + i];
    }
}

__global__ void diag_normalize(float *A, float *I, int n, int i) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < n && y < n)
    if (x == y && x == i) {
      I[x * n + y] /= A[i * n + i];
      A[x * n + y] /= A[i * n + i];
    }
}

__global__ void gaussjordan(float *A, float *I, int n, int i) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < n && y < n) {
    if (x != i) {
      I[x * n + y] -= I[i * n + y] * A[x * n + i];
      if (y != i) {
        A[x * n + y] -= A[i * n + y] * A[x * n + i];
      }
    }
  }
}

__global__ void set_zero(float *A, float *I, int n, int i) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < n && y < n) {
    if (x != i) {
      if (y == i) {
        A[x * n + y] = 0;
      }
    }
  }
}

void savetofile(float *A, string s, int n, int h) {
  std::ofstream plik;
  plik.open(s);

  for (int j = 0; j < h; j++) {
    for (int i = 0; i < h; i++) {
      plik << A[j * n + i] << "\t";
    }
    plik << endl;
  }
  plik.close();
}

int main() {
  const int n = 100;
  // creating input
  float *iL = new float[n * n];
  float *L = new float[n * n];
  matrix_read(L, n);

  cout << "inv\n";
  float *d_A, *d_L, *I, *dI;
  int ddsize = n * n * sizeof(float);

  dim3 threadsPerBlock(blocksize, blocksize);
  dim3 numBlocks((n + blocksize - 1) / blocksize,
                 (n + blocksize - 1) / blocksize);
  // memory allocation
  hipMalloc((void **)&d_A, ddsize);
  hipMalloc((void **)&dI, ddsize);
  I = new float[n * n];

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (i == j)
        I[i * n + i] = 1.0;
      else
        I[i * n + j] = 0.0;
    }
  }

  // copy data from CPU to GPU
  hipMemcpy(d_A, L, ddsize, hipMemcpyHostToDevice);
  hipMemcpy(dI, I, ddsize, hipMemcpyHostToDevice);

  // L^(-1)
  for (int i = 0; i < n; i++) {
    nodiag_normalize<<<numBlocks, threadsPerBlock>>>(d_A, dI, n, i);
    diag_normalize<<<numBlocks, threadsPerBlock>>>(d_A, dI, n, i);
    gaussjordan<<<numBlocks, threadsPerBlock>>>(d_A, dI, n, i);
    set_zero<<<numBlocks, threadsPerBlock>>>(d_A, dI, n, i);
  }

  // copy data from GPU to CPU
  hipMemcpy(iL, dI, ddsize, hipMemcpyDeviceToHost);
  hipMemcpy(I, d_A, ddsize, hipMemcpyDeviceToHost);

  savetofile(iL, "inv.txt", n, n);
  // savetofile(I, "I.txt", n, n);
  hipFree(d_A);
  hipFree(dI);

  float *c = new float[n * n];
  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++) {
      c[i * n + j] = 0;  // put the initial value to zero
      for (int x = 0; x < n; x++)
        c[i * n + j] = c[i * n + j] +
                       L[i * n + x] * iL[x * n + j];  // matrix multiplication
    }
  savetofile(c, "c.txt", n, n);

  delete[] I;
  delete[] L;
  delete[] iL;

  return 0;
}