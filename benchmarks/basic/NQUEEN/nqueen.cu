// N-queen for CUDA
// origin:
// https://github.com/charitha22/cgo22ae-darm-benchmarks/tree/main/benchmarks/NQU
// Copyright(c) 2008 Ping-Che Chen


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define THREAD_NUM 96

/* -------------------------------------------------------------------
 * This is a non-recursive version of n-queen backtracking solver.
 * This provides the basis for the CUDA version.
 * -------------------------------------------------------------------
 */

long long solve_nqueen(int n) {
  unsigned int mask[32];
  unsigned int l_mask[32];
  unsigned int r_mask[32];
  unsigned int m[32];

  if (n <= 0 || n > 32) {
    return 0;
  }

  const unsigned int t_mask = (1 << n) - 1;
  long long total = 0;
  long long upper_total = 0;
  int i = 0, j;
  unsigned int index;

  mask[0] = 0;
  l_mask[0] = 0;
  r_mask[0] = 0;
  m[0] = 0;

  for (j = 0; j < (n + 1) / 2; j++) {
    index = (1 << j);
    m[0] |= index;

    mask[1] = index;
    l_mask[1] = index << 1;
    r_mask[1] = index >> 1;
    m[1] = (mask[1] | l_mask[1] | r_mask[1]);
    i = 1;

    if (n % 2 == 1 && j == (n + 1) / 2 - 1) {
      upper_total = total;
      total = 0;
    }

    while (i > 0) {
      if ((m[i] & t_mask) == t_mask) {
        i--;
      } else {
        index = ((m[i] + 1) ^ m[i]) & ~m[i];
        m[i] |= index;
        if ((index & t_mask) != 0) {
          if (i + 1 == n) {
            total++;
            i--;
          } else {
            mask[i + 1] = mask[i] | index;
            l_mask[i + 1] = (l_mask[i] | index) << 1;
            r_mask[i + 1] = (r_mask[i] | index) >> 1;
            m[i + 1] = (mask[i + 1] | l_mask[i + 1] | r_mask[i + 1]);
            i++;
          }
        } else {
          i--;
        }
      }
    }
  }

  if (n % 2 == 0) {
    return total * 2;
  } else {
    return upper_total * 2 + total;
  }
}

/* --------------------------------------------------------------------------
 * This is a non-recursive version of n-queen backtracking solver for CUDA.
 * It receives multiple initial conditions from a CPU iterator, and count
 * each conditions.
 * --------------------------------------------------------------------------
 */

__global__ void solve_nqueen_cuda_kernel(
    int n, int mark, unsigned int* total_masks, unsigned int* total_l_masks,
    unsigned int* total_r_masks, unsigned int* results, int total_conditions) {
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int idx = bid * blockDim.x + tid;

  __shared__ unsigned int mask[THREAD_NUM][10];
  __shared__ unsigned int l_mask[THREAD_NUM][10];
  __shared__ unsigned int r_mask[THREAD_NUM][10];
  __shared__ unsigned int m[THREAD_NUM][10];

  __shared__ unsigned int sum[THREAD_NUM];

  const unsigned int t_mask = (1 << n) - 1;
  int total = 0;
  int i = 0;
  unsigned int index;

  if (idx < total_conditions) {
    mask[tid][i] = total_masks[idx];
    l_mask[tid][i] = total_l_masks[idx];
    r_mask[tid][i] = total_r_masks[idx];
    m[tid][i] = mask[tid][i] | l_mask[tid][i] | r_mask[tid][i];

    while (i >= 0) {
      if ((m[tid][i] & t_mask) == t_mask) {
        i--;
      } else {
        index = (m[tid][i] + 1) & ~m[tid][i];
        m[tid][i] |= index;
        if ((index & t_mask) != 0) {
          if (i + 1 == mark) {
            total++;
            i--;
          } else {
            mask[tid][i + 1] = mask[tid][i] | index;
            l_mask[tid][i + 1] = (l_mask[tid][i] | index) << 1;
            r_mask[tid][i + 1] = (r_mask[tid][i] | index) >> 1;
            m[tid][i + 1] =
                (mask[tid][i + 1] | l_mask[tid][i + 1] | r_mask[tid][i + 1]);
            i++;
          }
        } else {
          i--;
        }
      }
    }

    sum[tid] = total;
  } else {
    sum[tid] = 0;
  }

  __syncthreads();

  // reduction
  if (tid < 64 && tid + 64 < THREAD_NUM) {
    sum[tid] += sum[tid + 64];
  }
  __syncthreads();
  if (tid < 32) {
    sum[tid] += sum[tid + 32];
  }
  __syncthreads();
  if (tid < 16) {
    sum[tid] += sum[tid + 16];
  }
  __syncthreads();
  if (tid < 8) {
    sum[tid] += sum[tid + 8];
  }
  __syncthreads();
  if (tid < 4) {
    sum[tid] += sum[tid + 4];
  }
  __syncthreads();
  if (tid < 2) {
    sum[tid] += sum[tid + 2];
  }
  __syncthreads();
  if (tid < 1) {
    sum[tid] += sum[tid + 1];
  }
  __syncthreads();

  if (tid == 0) {
    results[bid] = sum[0];
  }
}

long long solve_nqueen_cuda(int n, int steps) {
  // generating start conditions
  unsigned int mask[32];
  unsigned int l_mask[32];
  unsigned int r_mask[32];
  unsigned int m[32];
  unsigned int index;

  if (n <= 0 || n > 32) {
    return 0;
  }

  unsigned int* total_masks = new unsigned int[steps];
  unsigned int* total_l_masks = new unsigned int[steps];
  unsigned int* total_r_masks = new unsigned int[steps];
  unsigned int* results = new unsigned int[steps];

  unsigned int* masks_cuda;
  unsigned int* l_masks_cuda;
  unsigned int* r_masks_cuda;
  unsigned int* results_cuda;

  hipMalloc((void**)&masks_cuda, sizeof(int) * steps);
  hipMalloc((void**)&l_masks_cuda, sizeof(int) * steps);
  hipMalloc((void**)&r_masks_cuda, sizeof(int) * steps);
  hipMalloc((void**)&results_cuda, sizeof(int) * steps / THREAD_NUM);

  const unsigned int t_mask = (1 << n) - 1;
  const unsigned int mark = n > 11 ? n - 10 : 2;
  long long total = 0;
  int total_conditions = 0;
  int i = 0, j;

  mask[0] = 0;
  l_mask[0] = 0;
  r_mask[0] = 0;
  m[0] = 0;

  bool computed = false;

  for (j = 0; j < n / 2; j++) {
    index = (1 << j);
    m[0] |= index;

    mask[1] = index;
    l_mask[1] = index << 1;
    r_mask[1] = index >> 1;
    m[1] = (mask[1] | l_mask[1] | r_mask[1]);
    i = 1;

    while (i > 0) {
      if ((m[i] & t_mask) == t_mask) {
        i--;
      } else {
        index = (m[i] + 1) & ~m[i];
        m[i] |= index;
        if ((index & t_mask) != 0) {
          mask[i + 1] = mask[i] | index;
          l_mask[i + 1] = (l_mask[i] | index) << 1;
          r_mask[i + 1] = (r_mask[i] | index) >> 1;
          m[i + 1] = (mask[i + 1] | l_mask[i + 1] | r_mask[i + 1]);
          i++;
          if (i == mark) {
            total_masks[total_conditions] = mask[i];
            total_l_masks[total_conditions] = l_mask[i];
            total_r_masks[total_conditions] = r_mask[i];
            total_conditions++;
            if (total_conditions == steps) {
              if (computed) {
                hipMemcpy(results, results_cuda,
                           sizeof(int) * steps / THREAD_NUM,
                           hipMemcpyDeviceToHost);

                for (int j = 0; j < steps / THREAD_NUM; j++) {
                  total += results[j];
                }

                computed = false;
              }

              // start computation
              hipMemcpy(masks_cuda, total_masks,
                         sizeof(int) * total_conditions,
                         hipMemcpyHostToDevice);
              hipMemcpy(l_masks_cuda, total_l_masks,
                         sizeof(int) * total_conditions,
                         hipMemcpyHostToDevice);
              hipMemcpy(r_masks_cuda, total_r_masks,
                         sizeof(int) * total_conditions,
                         hipMemcpyHostToDevice);

              solve_nqueen_cuda_kernel<<<steps / THREAD_NUM, THREAD_NUM>>>(
                  n, n - mark, masks_cuda, l_masks_cuda, r_masks_cuda,
                  results_cuda, total_conditions);

              computed = true;

              total_conditions = 0;
            }
            i--;
          }
        } else {
          i--;
        }
      }
    }
  }

  if (computed) {
    hipMemcpy(results, results_cuda, sizeof(int) * steps / THREAD_NUM,
               hipMemcpyDeviceToHost);

    for (int j = 0; j < steps / THREAD_NUM; j++) {
      total += results[j];
    }

    computed = false;
  }

  hipMemcpy(masks_cuda, total_masks, sizeof(int) * total_conditions,
             hipMemcpyHostToDevice);
  hipMemcpy(l_masks_cuda, total_l_masks, sizeof(int) * total_conditions,
             hipMemcpyHostToDevice);
  hipMemcpy(r_masks_cuda, total_r_masks, sizeof(int) * total_conditions,
             hipMemcpyHostToDevice);

  solve_nqueen_cuda_kernel<<<steps / THREAD_NUM, THREAD_NUM>>>(
      n, n - mark, masks_cuda, l_masks_cuda, r_masks_cuda, results_cuda,
      total_conditions);

  hipMemcpy(results, results_cuda, sizeof(int) * steps / THREAD_NUM,
             hipMemcpyDeviceToHost);

  for (int j = 0; j < steps / THREAD_NUM; j++) {
    total += results[j];
  }

  total *= 2;

  if (n % 2 == 1) {
    computed = false;
    total_conditions = 0;

    index = (1 << (n - 1) / 2);
    m[0] |= index;

    mask[1] = index;
    l_mask[1] = index << 1;
    r_mask[1] = index >> 1;
    m[1] = (mask[1] | l_mask[1] | r_mask[1]);
    i = 1;

    while (i > 0) {
      if ((m[i] & t_mask) == t_mask) {
        i--;
      } else {
        index = (m[i] + 1) & ~m[i];
        m[i] |= index;
        if ((index & t_mask) != 0) {
          mask[i + 1] = mask[i] | index;
          l_mask[i + 1] = (l_mask[i] | index) << 1;
          r_mask[i + 1] = (r_mask[i] | index) >> 1;
          m[i + 1] = (mask[i + 1] | l_mask[i + 1] | r_mask[i + 1]);
          i++;
          if (i == mark) {
            total_masks[total_conditions] = mask[i];
            total_l_masks[total_conditions] = l_mask[i];
            total_r_masks[total_conditions] = r_mask[i];
            total_conditions++;
            if (total_conditions == steps) {
              if (computed) {
                hipMemcpy(results, results_cuda,
                           sizeof(int) * steps / THREAD_NUM,
                           hipMemcpyDeviceToHost);

                for (int j = 0; j < steps / THREAD_NUM; j++) {
                  total += results[j];
                }

                computed = false;
              }

              // start computation
              hipMemcpy(masks_cuda, total_masks,
                         sizeof(int) * total_conditions,
                         hipMemcpyHostToDevice);
              hipMemcpy(l_masks_cuda, total_l_masks,
                         sizeof(int) * total_conditions,
                         hipMemcpyHostToDevice);
              hipMemcpy(r_masks_cuda, total_r_masks,
                         sizeof(int) * total_conditions,
                         hipMemcpyHostToDevice);

              solve_nqueen_cuda_kernel<<<steps / THREAD_NUM, THREAD_NUM>>>(
                  n, n - mark, masks_cuda, l_masks_cuda, r_masks_cuda,
                  results_cuda, total_conditions);

              computed = true;

              total_conditions = 0;
            }
            i--;
          }
        } else {
          i--;
        }
      }
    }

    if (computed) {
      hipMemcpy(results, results_cuda, sizeof(int) * steps / THREAD_NUM,
                 hipMemcpyDeviceToHost);

      for (int j = 0; j < steps / THREAD_NUM; j++) {
        total += results[j];
      }

      computed = false;
    }

    hipMemcpy(masks_cuda, total_masks, sizeof(int) * total_conditions,
               hipMemcpyHostToDevice);
    hipMemcpy(l_masks_cuda, total_l_masks, sizeof(int) * total_conditions,
               hipMemcpyHostToDevice);
    hipMemcpy(r_masks_cuda, total_r_masks, sizeof(int) * total_conditions,
               hipMemcpyHostToDevice);

    solve_nqueen_cuda_kernel<<<steps / THREAD_NUM, THREAD_NUM>>>(
        n, n - mark, masks_cuda, l_masks_cuda, r_masks_cuda, results_cuda,
        total_conditions);

    hipMemcpy(results, results_cuda, sizeof(int) * steps / THREAD_NUM,
               hipMemcpyDeviceToHost);

    for (int j = 0; j < steps / THREAD_NUM; j++) {
      total += results[j];
    }
  }

  hipFree(masks_cuda);
  hipFree(l_masks_cuda);
  hipFree(r_masks_cuda);
  hipFree(results_cuda);

  delete[] total_masks;
  delete[] total_l_masks;
  delete[] total_r_masks;
  delete[] results;

  return total;
}

int main(int argc, char** argv) {
  int n = 8;
  long long solution_cpu, solution_gpu;
  int argstart = 1, steps = 24576;

  if (argc < argstart + 1) {
    printf("Usage: %s n steps\n", argv[0]);
    printf("  n: n-queen\n");
    printf("  steps: step number for GPU\n");
    printf("Default to 8 queen\n");
  } else {
    n = atoi(argv[argstart]);
    if (n <= 1 || n > 32) {
      printf("Invalid n, n should be > 1 and <= 32\n");
      printf("Note: n > 18 will require a very very long time to compute!\n");
      return 0;
    }

    if (argc >= argstart + 2) {
      steps = atoi(argv[argstart + 1]);
      if (steps <= THREAD_NUM || steps % THREAD_NUM != 0) {
        printf("Invalid step, step should be multiple of %d\n", THREAD_NUM);
        return 0;
      }
    }
  }
  solution_cpu = solve_nqueen(n);
  solution_gpu = solve_nqueen_cuda(n, steps);

  if (solution_cpu != solution_gpu) {
    printf("Calculation Error!\n");
    return 1;
  }

  printf("Calculation right!\n");
  return 0;
}