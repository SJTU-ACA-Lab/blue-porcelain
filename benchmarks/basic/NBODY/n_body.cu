// origin: https://github.com/9prady9/CUDA/blob/master/nBodyComputation.cu

#include <hip/hip_runtime.h>
#include <malloc.h>
#include <math.h>
#include <stdio.h>
#include <time.h>

#define MAX_BLKS 65537
#define MAX_THRDS_P_BLK 512

struct Vector {
  float x;
  float y;
  float z;
  __device__ float d_influenceBy(Vector p) {
    return 1 / sqrt((x - p.x) * (x - p.x) + (y - p.y) * (y - p.y) +
                    (z - p.z) * (z - p.z));
  }
  __host__ float h_influenceBy(Vector p) {
    return 1 / sqrt((x - p.x) * (x - p.x) + (y - p.y) * (y - p.y) +
                    (z - p.z) * (z - p.z));
  }
};

__host__ int ciel(float value) {
  float mantissa = value - (int)value;
  return ((int)value + (mantissa == 0 ? 0 : 1));
}

/**
 * Kernel is launched with as many blocks as bodies. Not an optimal strategy but
 * first iteration. In each block, if number of threads is a power of two which
 * is computed based on number of bodies given Hence, the binary reduction in
 * kernel is much simpler since thread load is power of two. Each thread
 * computes forces of <bodiesPerThread> bodies on itself, followed by a
 * reduction to compute overall force.
 */
__global__ void forceComp(Vector *positions, int bodyCount,
                          float *resultantForce, int bodiesPerThread) {
  extern __shared__ float perBlockCache[];
  int tid = threadIdx.x * bodiesPerThread;
  int Limit = tid + bodiesPerThread;

  if (tid < bodyCount) {
    perBlockCache[threadIdx.x] = 0.0;
    while (tid < Limit) {
      if (blockIdx.x != tid)
        perBlockCache[threadIdx.x] +=
            positions[blockIdx.x].d_influenceBy(positions[tid]);
      tid++;
    }
    __syncthreads();

    /* now do reduction by addition for the resultant
     * force on body with Id = blockIdx.x */
    int reduceDim = blockDim.x / 2;
    while (reduceDim > 0) {
      if (threadIdx.x < reduceDim)
        perBlockCache[threadIdx.x] += perBlockCache[threadIdx.x + reduceDim];
      __syncthreads();
      reduceDim /= 2;
    }
    if (threadIdx.x == 0) resultantForce[blockIdx.x] = perBlockCache[0];
  }
}

/**
 * Program requires a numerical input i.e. the number
 * of bodies participating in the n-body simulation
 * This number should be less than MAX_BLKS = 65537
 */
int main(int argc, char *argv[]) {
  int host_bodyCount = 256;

  size_t res_size;
  int iter;
  float tempResult;
  Vector *host_positions;
  float *host_resultantForce;
  Vector *dev_positions;
  float *dev_resultantForce;

  size_t size = host_bodyCount * sizeof(Vector);
  int blocksPerGrid = host_bodyCount;
  int thrdCntHold = MAX_THRDS_P_BLK;
  for (int i = 1; host_bodyCount < thrdCntHold && thrdCntHold > 2; ++i)
    thrdCntHold >>= 1;
  int threadsPerBlock = thrdCntHold;
  int bodiesPerThread = ciel((float)host_bodyCount / threadsPerBlock);
  res_size = threadsPerBlock * sizeof(float);

  printf("Blocks per Grid: %d\nThreads per Block: %d\n", blocksPerGrid,
         threadsPerBlock);
  srand(time(NULL));

  /* Allocate host memory to prepare data */
  host_positions = (Vector *)malloc(size);
  host_resultantForce = (float *)malloc(res_size);
  for (iter = 0; iter < host_bodyCount; iter++) {
    host_positions[iter].x = iter + 1.0;
    host_positions[iter].y = iter + 1.0;
    host_positions[iter].z = iter + 1.0;
    // printf("Body %d position is (%f,%f,%f)\n", iter+1,
    // host_positions[iter].x, host_positions[iter].y, host_positions[iter].z
    // );
  }

  /* Allocate device memory, GPU memory */
  hipMalloc((void **)&dev_positions, size);
  hipMalloc((void **)&dev_resultantForce, res_size);

  /* Copy data from host to device */
  hipMemcpy(dev_positions, host_positions, size, hipMemcpyHostToDevice);
  forceComp<<<blocksPerGrid, threadsPerBlock, res_size>>>(
      dev_positions, host_bodyCount, dev_resultantForce, bodiesPerThread);

  /* Copy result from device to host */
  hipMemcpy(host_resultantForce, dev_resultantForce, res_size,
             hipMemcpyDeviceToHost);

  /* Compute on host for comparison */
  int error = 0;
  for (int i = 0; i < host_bodyCount; i++) {
    tempResult = 0.0;
    for (iter = 0; iter < host_bodyCount; iter++) {
      if (iter != i)
        tempResult =
            tempResult + host_positions[i].h_influenceBy(host_positions[iter]);
    }
    printf("Force on me(%d) is %.3f; Device result is %.3f\n", i + 1,
           tempResult, host_resultantForce[i]);
    if (abs((tempResult - host_resultantForce[i]) / host_resultantForce[i]) >
        1e-3)
      error = 1;
  }

  /* clear all memory  */
  hipFree(dev_positions);
  hipFree(dev_resultantForce);
  free(host_positions);
  free(host_resultantForce);

  if (error == 1) {
    printf("Calculation Error!\n");
    return 1;
  }

  printf("Calculation right!\n");
  return 0;
}