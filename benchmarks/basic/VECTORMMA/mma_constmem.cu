// Copyright 2022, ACALab of SJTU


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 512

void host_add(int *a, int *b, int *c) {
  for (int idx = 0; idx < N; idx++) c[idx] = a[idx] + b[idx];
}
__constant__ int d[N] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
__global__ void device_add(int *a, int *b, int *c) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < N) {
    c[index] = a[index] * b[index] + d[index];
  }
}

// basically just fills the array with index.
void fill_array(int *data) {
  for (int i = 0; i < N; ++i) {
    data[i] = rand() % (int)1000;
  }
}

void print_output(int *a, int *b, int *c) {
  for (int idx = 0; idx < N; idx++)
    printf("\n %d + %d  = %d", a[idx], b[idx], c[idx]);
}
int main(void) {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;  // device copies of a, b, c
  int threads_per_block = 0, no_of_blocks = 0;

  int size = N * sizeof(int);

  // Alloc space for host copies of a, b, c and setup input values
  a = (int *)malloc(size);
  fill_array(a);
  b = (int *)malloc(size);
  fill_array(b);
  c = (int *)malloc(size);

  // Alloc space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  threads_per_block = 256;
  no_of_blocks = N / threads_per_block;
  device_add<<<no_of_blocks, threads_per_block>>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  int flag = 0;
  for (int i = 0; i < N; i++) {
    if (i < 10) {
      if (c[i] != a[i] * b[i] + i + 1) {
        printf("\nCalculation Error!\n");
        flag = 1;
        break;
      }
    } else {
      if (c[i] != a[i] * b[i]) {
        printf("\nCalculation Error!\n");
        flag = 1;
        break;
      }
    }
  }

  free(a);
  free(b);
  free(c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  if (flag == 0) {
    printf("Calculation right!\n");
    return 0;
  }

  return 1;
}
