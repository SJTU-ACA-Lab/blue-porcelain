// Copyright 2022, ACALab of SJTU


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define imin(a, b) (a < b ? a : b)

const int N = 16 * 1024;

const int threadsPerBlock = 256;

const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(float *a, float *b, float *c) {
  __shared__ float cache[threadsPerBlock];  // 线程块共享
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIndex = threadIdx.x;

  float temp = 0;
  while (tid < N) {
    temp += a[tid] * b[tid];
    tid += blockDim.x * gridDim.x;
  }

  // 设置cache中相应位置上的值
  cache[cacheIndex] = temp;

  // 对线程块内线程进行同步
  __syncthreads();

  // 规约运算，求和
  int i = blockDim.x / 2;
  while (i != 0) {
    if (cacheIndex < i) cache[cacheIndex] += cache[cacheIndex + i];
    __syncthreads();
    i /= 2;
  }
  if (cacheIndex == 0) c[blockIdx.x] = cache[0];
}

int main() {
  float *a, *b, c, *partial_c;
  float *dev_a, *dev_b, *dev_partial_c;

  // 在CPU上分配内存
  a = (float *)malloc(N * sizeof(float));
  b = (float *)malloc(N * sizeof(float));
  partial_c = (float *)malloc(blocksPerGrid * sizeof(float));

  // 在GPU上分配内存
  (hipMalloc((void **)&dev_a, N * sizeof(float)));
  (hipMalloc((void **)&dev_b, N * sizeof(float)));
  (hipMalloc((void **)&dev_partial_c, blocksPerGrid * sizeof(float)));

  // 填充主机内存
  for (int i = 0; i < N; i++) {
    a[i] = i;
    b[i] = i * 2;
  }

  // 将数组a和b复制到GPU
  (hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice));
  (hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice));

  dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);

  // 将数组c从GPU复制到CPU
  (hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float),
              hipMemcpyDeviceToHost));

  // 在CPU上完成最终的求和运算
  c = 0;
  for (int i = 0; i < blocksPerGrid; i++) {
    c += partial_c[i];
  }

#define sum_squares(x) (x * (x + 1) * (2 * x + 1) / 6)
  float cpu_value = 2 * sum_squares((float)(N - 1));
  printf("Does GPU value %.6g = %.6g?\n", c, cpu_value);

  // 释放GPU上的内存
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_partial_c);

  // 释放CPU上的内存
  free(a);
  free(b);
  free(partial_c);

  if ((c - cpu_value) / cpu_value < 0.00001 &&
      (c - cpu_value) / cpu_value > -0.00001) {
    std::cout << "Calculation Right!!\n";
    return 0;
  } else {
    std::cout << "Calculation Error!\n";
  }

  return 1;
}