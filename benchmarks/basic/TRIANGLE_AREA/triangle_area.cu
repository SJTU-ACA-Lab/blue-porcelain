// Copyright 2022, ACALab of SJTU


#include <hip/hip_runtime.h>
#include <math.h>
#include <random>
#include <stdio.h>
#include <stdlib.h>

#define N 2048

void host_triangle_area(float *arr1, float *arr2, float *arr3, float *res) {
  float a, b, c;
  for (int idx = 0; idx < N; idx++) {
    a = arr1[idx];
    b = arr2[idx];
    c = arr2[idx];
    if (a + b > c && a + c > b && b + c > a) {
      float p = (a + b + c) * 0.5;  //计算半周长
      float area = sqrt(p * (p - a) * (p - b) * (p - c));
      res[idx] = area;

    } else {
      res[idx] = 0;
    }
  }
}

__global__ void gpu_triangle_area(float *arr1, float *arr2, float *arr3,
                                  float *res) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  float a, b, c;
  a = arr1[idx];
  b = arr2[idx];
  c = arr2[idx];
  if (a + b > c && a + c > b && b + c > a) {
    float p = (a + b + c) * 0.5;  //计算半周长
    float area = sqrt(p * (p - a) * (p - b) * (p - c));
    res[idx] = area;

  } else {
    res[idx] = 0;
  }
}

int main(void) {
  float *a, *b, *c, *res, *cpu_res;
  float *d_a, *d_b, *d_c, *d_res;  // device copies of a, b, c
  int threads_per_block = 0, no_of_blocks = 0;

  int size = N * sizeof(float);

  std::random_device rd;
  std::default_random_engine eng(rd());
  std::uniform_real_distribution<float> distr;

  // Alloc space for host copies of a, b, c and setup input values
  a = (float *)malloc(size);
  for (int i = 0; i < N; i++) {
    a[i] = distr(eng);
  }
  b = (float *)malloc(size);
  for (int i = 0; i < N; i++) {
    b[i] = distr(eng);
  }
  c = (float *)malloc(size);
  for (int i = 0; i < N; i++) {
    c[i] = distr(eng);
  }
  res = (float *)malloc(size);
  cpu_res = (float *)malloc(size);

  // Alloc space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  hipMalloc((void **)&d_res, size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

  threads_per_block = 256;
  no_of_blocks = (N + threads_per_block - 1) / threads_per_block;
  gpu_triangle_area<<<no_of_blocks, threads_per_block>>>(d_a, d_b, d_c, d_res);

  // Copy result back to host
  hipMemcpy(res, d_res, size, hipMemcpyDeviceToHost);

  host_triangle_area(a, b, c, cpu_res);

  int flag = 0;
  for (int i = 0; i < N; i++) {
    if (abs((cpu_res[i] - res[i]) / cpu_res[i]) > 1e-3) {
      printf("C[%d]:%f != %f\n", i, cpu_res[i], res[i]);
      flag = 1;
    }
  }

  free(a);
  free(b);
  free(c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  if (flag == 0) {
    printf("Calculation right!\n");
    return 0;
  }

  return 1;
}