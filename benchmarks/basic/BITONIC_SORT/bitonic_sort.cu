// Copyright 2022, ACALab of SJTU


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE (1 << 6)
#endif

#define GRID_SIZE ((1 << 10) / BLOCK_SIZE)

__device__ inline void swap(int &a, int &b) {
  unsigned int tmp = a;
  a = b;
  b = tmp;
}

__global__ static void bitonicSort(int *values) {
  extern __shared__ int shared[];
  const unsigned int tid = threadIdx.x;
  int offset = blockIdx.x * BLOCK_SIZE;
  shared[tid] = values[offset + tid];

  __syncthreads();

  for (unsigned int k = 2; k <= BLOCK_SIZE; k *= 2) {
    for (unsigned int j = k / 2; j > 0; j /= 2) {
      unsigned int ixj = tid ^ j;
      if (ixj > tid) {
        if ((tid & k) == 0) {
          if (shared[ixj] < shared[tid]) swap(shared[tid], shared[ixj]);
        }

        else {
          if (shared[ixj] > shared[tid]) swap(shared[tid], shared[ixj]);
        }
      }
      __syncthreads();
    }
  }
  values[offset + tid] = shared[tid];
}

void initialize_data(int *data, int size) {
  for (int i = 0; i < size; i++) data[i] = BLOCK_SIZE - i;
}

int main(int argc, char **argv) {
  srand(0);
  int *values_h, *values_d;
  int size = BLOCK_SIZE * GRID_SIZE;

  values_h = (int *)malloc(sizeof(int) * size);
  hipMalloc((void **)&values_d, sizeof(int) * size);

  initialize_data(values_h, size);

  hipMemcpy(values_d, values_h, sizeof(int) * size, hipMemcpyHostToDevice);
  bitonicSort<<<GRID_SIZE, BLOCK_SIZE, sizeof(int) * BLOCK_SIZE>>>(values_d);
  hipMemcpy(values_h, values_d, sizeof(int) * size, hipMemcpyDeviceToHost);

  bool pass = true;

  for (int blk = 0; blk < GRID_SIZE; ++blk) {
    int offset = blk * BLOCK_SIZE;
    for (int i = offset; i < offset + BLOCK_SIZE - 1; i++) {
      if (values_h[i] > values_h[i + 1]) {
        pass = false;
        break;
      }
    }
    if (!pass) {
      printf("Calculation Error!\n");
      break;
    }
  }

  hipFree(values_d);
  free(values_h);

  if (pass) {
    printf("Calculation right!\n");
    return 0;
  }

  return 1;
}
