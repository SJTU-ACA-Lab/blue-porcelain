// Copyright 2022, ACALab of SJTU


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N 2048

void host_div(float *a, float *b, float *c) {
  for (int idx = 0; idx < N; idx++) c[idx] = a[idx] + b[idx];
}

__global__ void vectorDiv(float *a, float *b, float *c) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  c[index] = a[index] / b[index];
}

// basically just fills the array with index.
void fill_array(float *data, float d) {
  for (int idx = 0; idx < N; idx++) data[idx] = (idx + d) / (d + 1);
}

int main(void) {
  float *a, *b, *c;
  float *d_a, *d_b, *d_c;  // device copies of a, b, c
  int threads_per_block = 0, no_of_blocks = 0;

  int size = N * sizeof(float);

  // Alloc space for host copies of a, b, c and setup input values
  a = (float *)malloc(size);
  fill_array(a, 3.0);
  b = (float *)malloc(size);
  fill_array(b, 5.0);
  c = (float *)malloc(size);

  // Alloc space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  threads_per_block = 256;
  no_of_blocks = (N + threads_per_block - 1) / threads_per_block;
  vectorDiv<<<no_of_blocks, threads_per_block>>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  int flag = 0;
  for (int i = 0; i < N; i++) {
    if (c[i] - (a[i] / b[i]) < -0.000001 || c[i] - (a[i] / b[i]) > 0.000001) {
      printf("C[%d]:%f != %f\n", i, c[i], a[i] / b[i]);
      flag = 1;
    }
  }

  free(a);
  free(b);
  free(c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  if (flag == 0) {
    printf("Calculation right!\n");
    return 0;
  }

  return 1;
}