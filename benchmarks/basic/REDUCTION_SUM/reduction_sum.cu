// Copyright 2022, ACALab of SJTU


#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

using namespace std;

const int threadsPerBlock = 512;
const int N = (1 << 11) - 3;
const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

__global__ void sum(float* arr, float* out, int N) {
  __shared__ float s_data[threadsPerBlock];
  unsigned int tid = threadIdx.x;
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < N) {
    s_data[tid] = arr[i];
  }
  __syncthreads();

  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s && i + s < N) {
      s_data[tid] += s_data[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0) {
    out[blockIdx.x] = s_data[0];
  }
}

int varifyOutput(float* predict, float* arr, int N) {
  float pred = 0.0;
  for (int i = 0; i < blocksPerGrid; i++) {
    pred += predict[i];
  }

  float result = 0.0;
  for (int i = 0; i < N; i++) {
    result += arr[i];
  }

  if (abs((pred - result) / result) > 1e-5) return 1;
  return 0;
}

int main() {
  float *a_host, *r_host;
  float *a_device, *r_device;

  hipHostMalloc(&a_host, N * sizeof(float), hipHostMallocDefault);
  hipHostMalloc(&r_host, blocksPerGrid * sizeof(float), hipHostMallocDefault);

  hipMalloc(&a_device, N * sizeof(float));
  hipMalloc(&r_device, blocksPerGrid * sizeof(float));

  std::random_device rd;
  std::default_random_engine eng(rd());
  std::normal_distribution<float> distr(-8.8, 8.8);

  for (int i = 0; i < N; i++) {
    a_host[i] = distr(eng);
  }
  for (int i = 0; i < blocksPerGrid; i++) {
    r_host[i] = 0.0;
  }

  hipMemcpyAsync(a_device, a_host, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpyAsync(r_device, r_host, blocksPerGrid * sizeof(float),
                  hipMemcpyHostToDevice);

  sum<<<blocksPerGrid, threadsPerBlock, 0>>>(a_device, r_device, N);

  hipMemcpy(r_host, r_device, blocksPerGrid * sizeof(float),
             hipMemcpyDeviceToHost);

  int error = varifyOutput(r_host, a_host, N);

  hipFree(r_device);
  hipFree(a_device);
  hipHostFree(r_host);
  hipHostFree(a_host);

  if (error) {
    cout << "Calculation Error!\n";
    return 1;
  }
  cout << "Calculation right!\n";
  return 0;
}