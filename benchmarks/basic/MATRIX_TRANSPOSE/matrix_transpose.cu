// Copyright 2022, ACALab of SJTU


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK 16

__global__ void parallelTransposeMemCoalescing(int* A, int* B, int m, int n) {
  __shared__ int block[BLOCK][BLOCK];

  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < m && j < n) {
    block[threadIdx.y][threadIdx.x] = A[i * n + j];
    __syncthreads();
    B[j * m + i] = block[threadIdx.y][threadIdx.x];
  }
}

int main(int argc, char* argv[]) {
  int m = 1024;
  int n = 2048;

  int* A = (int*)malloc(m * n * sizeof(int));
  int* B = (int*)malloc(m * n * sizeof(int));

  int i;
  for (i = 0; i < m * n; ++i) A[i] = rand() % 100;

  int *d_A, *d_B;
  hipMalloc(&d_A, n * m * sizeof(int));
  hipMalloc(&d_B, n * m * sizeof(int));

  // dimensions
  dim3 threadblock(BLOCK, BLOCK);
  dim3 grid(1 + n / threadblock.x, 1 + m / threadblock.y);

  // copying A to the GPU
  hipMemcpy(d_A, A, n * m * sizeof(int), hipMemcpyHostToDevice);

  // calling function
  parallelTransposeMemCoalescing<<<grid, threadblock>>>(d_A, d_B, m, n);

  // once the function has been called I copy the result in matrix
  hipMemcpy(B, d_B, n * m * sizeof(int), hipMemcpyDeviceToHost);

  /////////////////////////// CHECKING RESULTS ///////////////////////////
  int error = 0;
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < m; ++j) {
      if (B[j * m + i] != A[i * n + j]) {
        error = 1;
        break;
      }
    }
  }

  hipFree(d_A);
  hipFree(d_B);

  free(A);
  free(B);

  if (error) {
    printf("Calculation Error!\n");
    return 1;
  }

  printf("Calculation right!\n");
  return 0;
}